#include <stdio.h>
#include <stdlib.h>

//####  OTHER INCLUDES  #########
#include "CA_HOST.cuh"//		#
#include "utils.cuh"
//###############################
CA_HOST h_CA;
CA_GPU* d_CA;
/**
 * Parse the command line argument and consequently sets
 * the appropriate file paths for the parameters of the simulation.
 * @param argc
 * @param argv
 */
void hostInitialize(int argc, char *argv[]){
	//get options and file paths from command arguments
	struct CommandLine cmd;
	cmd.parseArgv(argc,argv);
	//configure hosts simulation
	h_CA.setDataFolderPath(cmd._load_path);
	h_CA.loadParameters(h_CA.s_parameters.c_str());
}


int main ( int argc, char *argv[] ){
	hipDeviceReset();
	hostInitialize(argc,argv);
	//configure CA HOST
	h_CA.simulationInit();

	h_CA.loadSubstates();
	h_CA.printParameters();

	d_CA=h_CA.deviceCAGPUInitialization();



	//host initialization and configuration completed
	h_CA.deviceMemoryFree(d_CA);
	//free CA_HOST memory
	h_CA.hostMemoryFree();
	printf("SIMULATION ENDED\n");
}
