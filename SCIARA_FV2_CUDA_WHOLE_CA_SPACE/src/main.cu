#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

//####  OTHER INCLUDES  #########
#include "CA_HOST.cuh"//		#
#include "utils.cuh"

//###############################

//####  GLOBAL VARIABLES  #########
CA_HOST h_CA;
CA_GPU* d_CA;

dim3 dimBlock;
dim3 dimGrid;
//###############################
/**
 * Parse the command line argument and consequently sets
 * the appropriate file paths for the parameters of the simulation.
 * @param argc
 * @param argv
 */
void hostInitialize(int argc, char *argv[]){
	//get options and file paths from command arguments
	struct CommandLine cmd;
	cmd.parseArgv(argc,argv);
	//configure hosts simulation
	h_CA.setDataFolderPath(cmd._load_path);
	h_CA.loadParameters();
}



//##### TRANSITION FUNCTION KERNELS ######

__global__ void printSubstateG(CA_GPU* d_CA, int substate){
	d_CA->printSubstate(substate);
}

/**
 * Lava emission from vents.
 * This kernel should be launched with wust one block in 1D
 * (on X dimension) and with the number of threads equals to the number of
 * vents (parameter numVents)
 * @param d_CA
 */
__global__ void emitLavaFromVents(CA_GPU* d_CA){

	if(blockIdx.x==0 && blockIdx.y==0){
		if(threadIdx.x < d_CA->numVents && threadIdx.y==0){
			d_CA->emitLavaFromVent(threadIdx.x);
		}
	}
}

/**
 * Temperature initialization
 * @param d_CA
 */
__global__ void temperatureInitialization(CA_GPU* d_CA){
	d_CA->cellTemperatureInitialize();
}

/**
 * Flows Computation
 * @param d_CA
 */
__global__ void computeFlows(CA_GPU* d_CA){
	d_CA->empiricalFlows();
}

/**
 * Flows reduction
 * It computes the new values of thicknes based on the previous step of
 * computation, hence the calculation of the outflows
 * It also compute the new lava temperature and handle the lava solidification
 * @param d_CA
 */
__global__ void reduceFlows(CA_GPU* d_CA){
	d_CA->distribuiteFlows();
}

/**
 * Temperture Update reduction
 * @param d_CA
 */
__global__ void temperatureUpdate(CA_GPU* d_CA){

}

/** NOT A KERNEL!
 * Copy matrix.
 * Hard Swap the content  current and updated matrix
 * Lanches a kernel in which thread explicitly copy the matrices from
 * the current to the updated version
 * @param d_CA
 */
// void copyMatricesMemCpyDevToDev(CA_GPU* d_CA){
//	h_CA->copyMatricesMemCpyDevToDev();
//}

/**
 * Copy matrix.
 * Hard Swap the content  current and updated matrix
 * Lanches a kernel in which thread explicitly copy the matrices from
 * the current to the updated version
 * @param d_CA
 */
__global__ void copyMatrices(CA_GPU* d_CA){
	d_CA->swapMatrices();
}


//#######################################
unsigned int nVents;
void globalTransitionFunction(){
	////kernel launch parameters settings
	dimBlock.x=8;
	dimBlock.y=8;
	computeKernelLaunchParameter(dimBlock.x,dimBlock.y,h_CA.getNr(),h_CA.getNc(),dimGrid);

//printSubstateG<<<dimGrid,dimBlock>>>(d_CA,THICKNESS);
	for(int i=0;i<10000;i++){
		emitLavaFromVents<<<1,nVents>>>(d_CA);
		temperatureInitialization<<<dimGrid,dimBlock>>>(d_CA);
		computeFlows<<<dimGrid,dimBlock>>>(d_CA);
		reduceFlows<<<dimGrid,dimBlock>>>(d_CA);
		//h_CA.copyMatricesMemCpyDevToDev();
		copyMatrices<<<dimGrid,dimBlock>>>(d_CA);
	}

}



int main ( int argc, char *argv[] ){
	hipDeviceReset();
	hostInitialize(argc,argv);
	//configure CA HOST
	h_CA.simulationInit();
	nVents = h_CA.getNumVents();

	h_CA.loadSubstates();
//	h_CA.printParameters();

	d_CA=h_CA.deviceCAGPUInitialization();


	/*
	 * GLOBAL TRANSITION FUNCTION ON GPU
	 */

	globalTransitionFunction();

	hipDeviceSynchronize();



	h_CA.copyBackFromGPU(d_CA);
	h_CA.saveSubstatesOnFile(h_CA.getDataFolder()+"/output/");

	//host initialization and configuration completed
	h_CA.deviceMemoryFree(d_CA);
	//free CA_HOST memory
	h_CA.hostMemoryFree();
	printf("SIMULATION ENDED\n");
}
